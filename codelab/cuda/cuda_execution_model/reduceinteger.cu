#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n) {
  unsigned int tid = threadIdx.x;
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int *idata = g_idata + blockIdx.x * blockDim.x;
  if (idx >= n) return;

  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    if ((tid % (2 * stride)) == 0) {
      idata[tid] += idata[tid + stride];
    }
    __syncthreads();
  }
  if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

int main(int argc, char **argv) {
  int dev = 0;
  hipDeviceProp_t device_prop;
  CHECK(hipGetDeviceProperties(&device_prop, dev));
  printf("%s starting reduction at ", argv[0]);
  printf(" device %d: %s", dev, device_prop.name);
  CHECK(hipSetDevice(dev));

  bool result = false;
  int size = 1 << 24;
  printf(" with array size %d", size);

  int blocksize = 1024;
  if (argc > 1) {
    blocksize = atoi(argv[1]);
  }
  dim3 block(blocksize, 1);
  dim3 grid((size + block.x -1)/ block.x, 1);
  printf("grid %d block %d \n", grid.x, block.x);
  
  size_t bytes = size * sizeof(int);
  int* h_idata = (int *)malloc(bytes);
  int* h_odata = (int *)malloc(grid.x * sizeof(int));
  int *tmp = (int *)malloc(bytes);

  for (int i = 0; i < size; ++i) {
    h_idata[i] = (int) (rand() & 0xFF);
  }

  memcpy(tmp, h_idata, bytes);

  double istart, ielaps;
  int gpu_sum = 0;
  int *d_idata = NULL;
  int *d_odata = NULL;

  CHECK(hipMalloc((void **)&d_idata, bytes));
  CHECK(hipMalloc((void **)&d_odata, grid.x * sizeof(int)));

  istart = seconds();
  int cpu_sum = 0;
  // int cpu_sum = recursiveReduce(tmp, size);
  ielaps = seconds() - istart;
  printf("cpu reduce elapsed %f sec cpu_sum %d\n", ielaps, cpu_sum);

  CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
  CHECK(hipDeviceSynchronize());
  istart = seconds();
  reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
  CHECK(hipDeviceSynchronize());
  ielaps = seconds() - istart;
  CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
  return EXIT_SUCCESS;

}
