#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char* argv[]) {
    int idev = 0;
    hipDeviceProp_t dev_prop;
    CHECK(hipGetDeviceProperties(&dev_prop, idev));

    printf("Device %d: %s \n", idev, dev_prop.name);
    printf("Number of multiprocessors: %d \n", dev_prop.multiProcessorCount);
    printf("Total amount of shapred memory per block: %4.2f kb \n", dev_prop.sharedMemPerBlock / 1024.0);
    printf("Total amount of const memory: %4.2f kb\n", dev_prop.totalConstMem/1024.0);
    printf("Total number of registers available per block: %d\n", dev_prop.regsPerBlock);
    printf("Warp size: %d\n", dev_prop.warpSize);
    printf("Maximum number of threads per block: %d\n", dev_prop.maxThreadsPerBlock);
    printf("Maximum number of threads per multiprocess: %d\n", dev_prop.maxThreadsPerMultiProcessor);
    printf("Maximum number of warps per multiprocessor: %d\n", dev_prop.maxThreadsPerMultiProcessor/32);

    return EXIT_SUCCESS;

}
