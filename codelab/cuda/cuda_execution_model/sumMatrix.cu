#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

void initialData(float* ip, const int size) {
    for (int i = 0; i < size; ++i) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny) {
    float *ia = A;
    float *ib = B;
    float *ic = C;
    for(int iy = 0; iy < ny; ++iy) {
        for(int ix = 0; ix < nx; ++ix) {
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx;
        ib += nx;
        ic += nx;
    }
    return;
}

void checkResult(float *host_ref, float *gpu_ref, const int N) {
    double epsilon = 1.0E-8;
    for(int i = 0; i < N; ++i) {
        if(abs(host_ref[i] - gpu_ref[i] > epsilon)) {
            printf("host %f gpu %f", host_ref[i], gpu_ref[i]);
            printf("Arrays do not match.\n");
        }
    }
}

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int nx, int ny) {
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny) {
        C[idx] = A[idx] + B[idx];
    }

}


int main(int argc, char* argv[]) {
    int dev = 0;
    hipDeviceProp_t device_prop;
    CHECK(hipGetDeviceProperties(&device_prop, dev));
    CHECK(hipSetDevice(dev));

    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    float *h_A, *h_B, *host_ref, *gpu_ref;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    host_ref = (float *)malloc(nBytes);
    gpu_ref = (float *)malloc(nBytes);
    
    float istart = seconds();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    float ielaps = seconds() - istart;

    memset(host_ref, 0, nBytes);
    memset(gpu_ref, 0, nBytes);
    istart = seconds();
    sumMatrixOnHost(h_A, h_B, host_ref, nx, ny);
    ielaps = seconds() - istart;

    float *d_MatA, *d_MatB, *d_MatC;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));
    CHECK(hipMalloc((void **)&d_MatB, nBytes));
    CHECK(hipMalloc((void **)&d_MatC, nBytes));

    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    if (argc > 2) {
      dimx = atoi(argv[1]);
      dimy = atoi(argv[0]);
    }
    dim3 block(dimx, dimy);
    dim3 grid((nx+block.x - 1/block.x), (ny + block.y - 1)/block.y);

    CHECK(hipDeviceSynchronize());
    istart = seconds();
    sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    CHECK(hipDeviceSynchronize());
    printf("sumMatrixOnGPU2D <<<(%d, %d), (%d, %d)>>> elapsed %f ms\n", grid.x, grid.y, block.x, block.y, ielaps);
    CHECK(hipGetLastError());
   
    CHECK(hipMemcpy(gpu_ref, d_MatC, nBytes, hipMemcpyDeviceToHost));
    checkResult(host_ref, gpu_ref, nxy);

    CHECK(hipFree(d_MatA));
    CHECK(hipFree(d_MatB));
    CHECK(hipFree(d_MatC));

    free(h_A);
    free(h_B);
    free(host_ref);
    free(gpu_ref);

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
