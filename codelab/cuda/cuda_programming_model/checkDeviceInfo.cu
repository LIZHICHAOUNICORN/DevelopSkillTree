#include <hip/hip_runtime.h>
#include <iostream>

#include "../common/common.h"

int main(int argc, char** argv) {

  printf("%s Starting .. \n", argv[0]);

  int device_count = 0;
  hipGetDeviceCount(&device_count);

  printf("Detected %d CUDA Capable device(s)\n", device_count);

  int dev = 0;
  int driver_version = 0;
  int runtime_version = 0;
  CHECK(hipSetDevice(dev));
  
  hipDeviceProp_t device_prop;
  CHECK(hipGetDeviceProperties(&device_prop, dev));
  printf("Device %d \"%s\"\n", dev, device_prop.name);

  hipDriverGetVersion(&driver_version);
  hipRuntimeGetVersion(&runtime_version);
  printf("CUDA Driver Version / Runtime Version %d, %d\n", driver_version, runtime_version);
  return 0;
}
