#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

void checkResult(float *host_ref, float *gpu_ref, const int N) {
  double epsilon = 1.0E-8;
  bool match = 1;
  for (int i = 0; i < N; ++i) {
    if(abs(host_ref[i] - gpu_ref[i]) > epsilon) {
      match = 0;
      printf("arrays do not match!\n");
      printf("host %5.2f gpu %5.2f at current: %d\n", host_ref[i], gpu_ref[i], i);
      break;
    }
  }
  if (match) printf("Arrays match!\n");
  return;
}

void InitialData(float* ip, int size) {
  time_t t;
  srand((unsigned) time(&t));
  for (int i = 0; i < size; i++) {
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
  return;
}

void SumArraysOnHost(float *a, float *b, float *c, const int N) {
  for (int idx = 0; idx < N; ++idx) {
    c[idx] = a[idx] + b[idx];
  }
}

__global__ void SumArraysOnGpu(float* a, float *b, float *c, const int N){
  int i = threadIdx.x;
  if (i < N) c[i] = a[i] + b[i];
}

int main(int argc, char **argv) {
  printf("%s Starting ..\n", argv[0]);
  
  int dev = 0;
  CHECK(hipSetDevice(dev));

  int n_elem = 1 << 5;
  printf("Vector size %d \n", n_elem);
  size_t n_bytes = n_elem * sizeof(float);

  float* h_a = (float*)malloc(n_bytes);
  float* h_b = (float*)malloc(n_bytes);
  float* host_ref = (float*)malloc(n_bytes);
  float* gpu_ref = (float*)malloc(n_bytes);

  InitialData(h_a, n_elem);
  InitialData(h_b, n_elem);

  memset(host_ref, 0, n_bytes);
  memset(gpu_ref, 0, n_bytes);

  float *d_a, *d_b, *d_c;
  CHECK(hipMalloc((float**)&d_a, n_bytes));
  CHECK(hipMalloc((float**)&d_b, n_bytes));
  CHECK(hipMalloc((float**)&d_c, n_bytes));

  CHECK(hipMemcpy(d_a, h_a, n_bytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_b, h_b, n_bytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_c, gpu_ref, n_bytes, hipMemcpyHostToDevice));
  
  dim3 block(n_elem);
  dim3 grid(1);
  SumArraysOnGpu<<<grid, block>>>(d_a, d_b, d_c, n_elem);
  printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);
  CHECK(hipMemcpy(gpu_ref, d_c, n_bytes, hipMemcpyDeviceToHost));
  SumArraysOnHost(h_a, h_b, host_ref, n_elem);

  checkResult(host_ref, gpu_ref, n_elem);
  CHECK(hipFree(d_a));
  CHECK(hipFree(d_b));
  CHECK(hipFree(d_c));

  free(h_a);
  free(h_b);
  free(host_ref);
  free(gpu_ref);

  CHECK(hipDeviceReset());
  return 0;

}
