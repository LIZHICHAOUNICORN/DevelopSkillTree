
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_world_GPU() {
  printf("hello world from GPU\n");
}

int main(int argc, char** argv) {
    printf("Hello world CPU\n");

    // Run kernel function.
    hello_world_GPU<<<1, 10>>>();

    // exiplicitly will explicitly destroy and clean up
    // all resources associated with current device 
    // with the current process.
    hipDeviceReset();
    return 0;
}
